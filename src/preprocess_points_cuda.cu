#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void generateVoxels_random_kernel(
    float *points, size_t points_size, float min_x_range, float max_x_range,
    float min_y_range, float max_y_range, float min_z_range, float max_z_range,
    float pillar_x_size, float pillar_y_size, float pillar_z_size,
    int grid_y_size, int grid_x_size, unsigned int *mask, float *voxels) {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx >= points_size)
        return;

    float4 point = ((float4 *)points)[point_idx];

    if (point.x < min_x_range || point.x >= max_x_range ||
        point.y < min_y_range || point.y >= max_y_range ||
        point.z < min_z_range || point.z >= max_z_range)
        return;

    int voxel_idx = floorf((point.x - min_x_range) / pillar_x_size);
    int voxel_idy = floorf((point.y - min_y_range) / pillar_y_size);
    unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

    unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

    if (point_id >= MAX_POINT_NUM_PER_VOXEL)
        return;
    float *address =
        voxels + (voxel_index * MAX_POINT_NUM_PER_VOXEL + point_id) * 4;
    atomicExch(address + 0, point.x);
    atomicExch(address + 1, point.y);
    atomicExch(address + 2, point.z);
    atomicExch(address + 3, point.w);
}

// 获取voxels [20000,32,4] mask存储每个voxel点数
hipError_t generateVoxels_random_launch(
    float *points, size_t points_size, float min_x_range, float max_x_range,
    float min_y_range, float max_y_range, float min_z_range, float max_z_range,
    float pillar_x_size, float pillar_y_size, float pillar_z_size,
    int grid_y_size, int grid_x_size, unsigned int *mask, float *voxels,
    hipStream_t stream) {
    int threadNum = 64;
    dim3 blocks((points_size + threadNum - 1) / threadNum);
    dim3 threads(threadNum);
    generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>(
        points, points_size, min_x_range, max_x_range, min_y_range, max_y_range,
        min_z_range, max_z_range, pillar_x_size, pillar_y_size, pillar_z_size,
        grid_y_size, grid_x_size, mask, voxels);

    // printf("hipMalloc function : %s\n",
    //    hipGetErrorString(hipGetLastError()));
    hipError_t err = hipGetLastError();
    return err;
}

__global__ void generateVoxelsList_kernel(
    float *points, size_t points_size, float min_x_range, float max_x_range,
    float min_y_range, float max_y_range, float min_z_range, float max_z_range,
    float pillar_x_size, float pillar_y_size, float pillar_z_size,
    int grid_y_size, int grid_x_size, unsigned int *mask, int *voxelsList) {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx >= points_size)
        return;

    float4 point = ((float4 *)points)[point_idx];

    if (point.x < min_x_range || point.x >= max_x_range ||
        point.y < min_y_range || point.y >= max_y_range ||
        point.z < min_z_range || point.z >= max_z_range) {
        voxelsList[point_idx] = -1;
        return;
    }

    int voxel_idx = floorf((point.x - min_x_range) / pillar_x_size);
    int voxel_idy = floorf((point.y - min_y_range) / pillar_y_size);
    unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;

    atomicAdd(&(mask[voxel_index]), 1);
    voxelsList[point_idx] = voxel_index;
}

__global__ void generateVoxels_kernel(float *points, size_t points_size,
                                      int *voxelsList, unsigned int *mask,
                                      float *voxels) {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx >= points_size)
        return;

    int voxel_index = voxelsList[point_idx];

    if (voxel_index == -1)
        return;
    int point_id = atomicAdd(&(mask[voxel_index]), 1);

    if (point_id >= MAX_POINT_NUM_PER_VOXEL)
        return;
    float *address =
        voxels + (voxel_index * MAX_POINT_NUM_PER_VOXEL + point_id) * 4;
    float4 point = ((float4 *)points)[point_idx];
    atomicExch(address + 0, point.x);
    atomicExch(address + 1, point.y);
    atomicExch(address + 2, point.z);
    atomicExch(address + 3, point.w);
}

__global__ void generateBaseFeatures_kernel(unsigned int *mask, float *voxels,
                                            int grid_y_size, int grid_x_size,
                                            unsigned int *pillar_num,
                                            float *voxel_features,
                                            unsigned int *voxel_point_num,
                                            unsigned int *voxel_coords) {
    unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (voxel_idx >= grid_x_size || voxel_idy >= grid_y_size)
        return;

    unsigned int voxel_index = voxel_idy * grid_x_size + voxel_idx;
    unsigned int count = mask[voxel_index];
    if (!(count > 0))
        return;
    count = count < MAX_POINT_NUM_PER_VOXEL ? count : MAX_POINT_NUM_PER_VOXEL;

    unsigned int current_pillarId = 0;
    current_pillarId = atomicAdd(pillar_num, 1);

    voxel_point_num[current_pillarId] = count;

    uint4 idx = {0, 0, voxel_idy, voxel_idx};
    ((uint4 *)voxel_coords)[current_pillarId] = idx;

    for (int i = 0; i < count; i++) {
        int inIndex = voxel_index * MAX_POINT_NUM_PER_VOXEL + i;
        int outIndex = current_pillarId * MAX_POINT_NUM_PER_VOXEL + i;
        ((float4 *)voxel_features)[outIndex] = ((float4 *)voxels)[inIndex];
    }

    // clear buffer for next infer
    atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(unsigned int *mask, float *voxels,
                                        int grid_y_size, int grid_x_size,
                                        unsigned int *pillar_num,
                                        float *voxel_features,
                                        unsigned int *voxel_point_num,
                                        unsigned int *voxel_coords,
                                        hipStream_t stream) {
    dim3 threads = {32, 32};
    dim3 blocks = {(grid_x_size + threads.x - 1) / threads.x,
                   (grid_y_size + threads.y - 1) / threads.y};

    generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>(
        mask, voxels, grid_y_size, grid_x_size, pillar_num, voxel_features,
        voxel_point_num, voxel_coords);
    hipError_t err = hipGetLastError();
    return err;
}

// 4 channels -> 10 channels
__global__ void
generateFeatures_kernel(float *voxel_features, unsigned int *voxel_point_num,
                        unsigned int *voxel_coords, unsigned int *voxel_count,
                        float voxel_x, float voxel_y, float voxel_z,
                        float range_min_x, float range_min_y, float range_min_z,
                        float *features) {
    // 4 * blockIdx.x +  threadIdx.x / 4
    int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x / WARP_SIZE;
    int point_idx = threadIdx.x % WARP_SIZE;

    int pillar_idx_inBlock = threadIdx.x / 32;
    unsigned int num_voxels = voxel_count[0];

    if (pillar_idx >= num_voxels)
        return;

    __shared__ float4 pillarSM[WARPS_PER_BLOCK][WARP_SIZE];
    __shared__ float4 pillarSumSM[WARPS_PER_BLOCK];
    __shared__ uint4 idxsSM[WARPS_PER_BLOCK];
    __shared__ int pointsNumSM[WARPS_PER_BLOCK];
    __shared__ float pillarOutSM[WARPS_PER_BLOCK][WARP_SIZE][FEATURES_SIZE];

    if (threadIdx.x < WARPS_PER_BLOCK) {
        pointsNumSM[threadIdx.x] =
            voxel_point_num[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
        idxsSM[threadIdx.x] =
            ((uint4 *)voxel_coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
        pillarSumSM[threadIdx.x] = {0, 0, 0, 0};
    }

    pillarSM[pillar_idx_inBlock][point_idx] =
        ((float4 *)voxel_features)[pillar_idx * WARP_SIZE + point_idx];
    __syncthreads();

    // calculate sm in a pillar
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
        atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x),
                  pillarSM[pillar_idx_inBlock][point_idx].x);
        atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y),
                  pillarSM[pillar_idx_inBlock][point_idx].y);
        atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z),
                  pillarSM[pillar_idx_inBlock][point_idx].z);
    }
    __syncthreads();

    // feature-mean
    float4 mean;
    float validPoints = pointsNumSM[pillar_idx_inBlock];
    mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
    mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
    mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

    mean.x = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
    mean.y = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
    mean.z = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;

    // calculate offset
    float x_offset =
        voxel_x / 2 + idxsSM[pillar_idx_inBlock].w * voxel_x + range_min_x;
    float y_offset =
        voxel_y / 2 + idxsSM[pillar_idx_inBlock].z * voxel_y + range_min_y;
    float z_offset =
        voxel_z / 2 + idxsSM[pillar_idx_inBlock].y * voxel_z + range_min_z;

    // feature-offset
    float4 center;
    center.x = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
    center.y = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
    center.z = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

    // store output
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
        pillarOutSM[pillar_idx_inBlock][point_idx][0] =
            pillarSM[pillar_idx_inBlock][point_idx].x;
        pillarOutSM[pillar_idx_inBlock][point_idx][1] =
            pillarSM[pillar_idx_inBlock][point_idx].y;
        pillarOutSM[pillar_idx_inBlock][point_idx][2] =
            pillarSM[pillar_idx_inBlock][point_idx].z;
        pillarOutSM[pillar_idx_inBlock][point_idx][3] =
            pillarSM[pillar_idx_inBlock][point_idx].w;

        pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
        pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
        pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

        pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
        pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;
        pillarOutSM[pillar_idx_inBlock][point_idx][9] = center.z;

    } else {
        pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

        pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

        pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
        pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    }

    __syncthreads();

    for (int i = 0; i < FEATURES_SIZE; i++) {
        int outputSMId = pillar_idx_inBlock * WARP_SIZE * FEATURES_SIZE +
                         i * WARP_SIZE + point_idx;
        int outputId =
            pillar_idx * WARP_SIZE * FEATURES_SIZE + i * WARP_SIZE + point_idx;
        features[outputId] = ((float *)pillarOutSM)[outputSMId];
    }
}

hipError_t
generateFeatures_launch(float *voxel_features, unsigned int *voxel_point_num,
                        unsigned int *voxel_coords, unsigned int *voxel_count,
                        float voxel_x, float voxel_y, float voxel_z,
                        float range_min_x, float range_min_y, float range_min_z,
                        float *features, hipStream_t stream) {
    dim3 blocks((MAX_VOXEL_NUM + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK);
    dim3 threads(WARPS_PER_BLOCK * WARP_SIZE); // 4 * 32

    generateFeatures_kernel<<<blocks, threads, 0, stream>>>(
        voxel_features, voxel_point_num, voxel_coords, voxel_count, voxel_x,
        voxel_y, voxel_z, range_min_x, range_min_y, range_min_z, features);

    hipError_t err = hipGetLastError();
    return err;
}
